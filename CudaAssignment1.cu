/*
* Kirin Sharma
* CS-425 Advanced Architecture
* CUDA Assignment 1
*
* This program creates and runs a CUDA kernel to modify a matrix based on a specific criteria
* in parallel using 1 block with 10x10 threads in the block.
*/

#include <hip/hip_runtime.h>
#include <iostream>

#include <stdlib.h>

using namespace std;

// CUDA kernel function to modify the matrix such that each element Mij = Mij * Vj
__global__ void modify(int *matrix, int *vector, int size)
{
    int row = threadIdx.y;
    int col = threadIdx.x;	
    int index = row * size + col;
	matrix[index] = matrix[index] * vector[col];
}

// Helper function to print a matrix
void printMatrix(int* matrix, int size) {
    for(int i = 0; i < size; i++) {
        for(int j = 0; j < size; j++) {
            cout << matrix[size * i + j] << "   ";
        }
        cout << "\n";
    }
}

// Helper function to print a vector
void printVector(int* vector, int size) {
    for(int i = 0; i < size; i++) {
        cout << vector[i] << "   ";
    }
}

int main()
{
    // Allocate unified memory for the (flattened) 2-D matrix and vector
	int size = 10;
	int *matrix;
	int *vector;
    hipMallocManaged(&matrix, size * size * sizeof(int));
    hipMallocManaged(&vector, size * sizeof(int));

    // Initialize the matrix and vector
    for(int i = 0; i < size; i++) {
        vector[i] = i + 1;
        for(int j = 0; j < size; j++) {
            matrix[i * size + j] = j + 1;
        }
    }

    // Print the original vector
    cout << "Original Vector:\n";
    printVector(vector, size);
    cout << "\n\n";

    // Print the original matrix
    cout << "Original Matrix:\n";
    printMatrix(matrix, size);
    cout << "\n\n";

    // Specify block dimension to be 10x10 threads and grid dimension to be 1x1
    dim3 gridDimension(1, 1);
    dim3 blockDimension(10, 10);

    // Launch the cuda kernel
    modify<<<gridDimension, blockDimension>>>(matrix, vector, size);
    hipDeviceSynchronize();

    // Verify matrix is modified correctly by printing to the screen
    // Expected to be the squares of the numbers 1-10 in each row
    cout << "Modified Matrix:\n";
    printMatrix(matrix, size);

    // Free cuda memory
    hipFree(matrix);
    hipFree(vector);

    return 0;

} // end main
